/* language: "cuda" 7 lines 4 code 2 comments 1 blanks */

// add vector
__host__ void add(const int* a, const int* b, int* c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
